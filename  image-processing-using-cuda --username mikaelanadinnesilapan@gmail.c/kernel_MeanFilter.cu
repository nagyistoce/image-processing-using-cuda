#include "hip/hip_runtime.h"
/*
**  Reduce noise by using Mean Filter with equal weights
*/

__global__ void MeanFilter ( uint *dst, int imageW, int imageH, int radius, float brightness, float contrast)
{
    const int ix = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const int iy = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    //Add half of a texel to always address exact texel centers
    const float x = (float)ix + 0.5f;
    const float y = (float)iy + 0.5f;

    if(ix < imageW && iy < imageH){

		float3 sum = {0,0,0};
		float4 fresult = {0,0,0,0};
		float count = 0.f;

        for( float i = -radius; i <= radius; i++)
            for( float j = -radius; j <= radius; j++)
            {
				fresult = tex2D(texImage, x + j, y + i);

				//adjust contrast
				float red = pow(fresult.x, contrast);
				float green = pow(fresult.y, contrast);
				float blue = pow(fresult.z, contrast);

				//adjust brightness
				red = red * (1.f - brightness) + brightness;
				green = green * (1.f - brightness) + brightness;
				blue = blue * (1.f - brightness) + brightness;

				sum.x += red;
				sum.y += green;
				sum.z += blue;
				count += 1.f;
            }

		sum.x /= count;
		sum.y /= count;
		sum.z /= count;

        //Write final result to global memory
        dst[imageW * iy + ix] = make_color(sum.x, sum.y, sum.z, 0);
    }
}

extern "C" void meanFilterWrapper (uint *dst, int imageW, int imageH, int radius, float brightness, float contrast) 
{
	//for more effective kernel execution
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	MeanFilter<<<grid, threads>>>(dst, imageW, imageH, radius, brightness, contrast);
}