#include "hip/hip_runtime.h"
__global__ void GrayErosion (unsigned int *dst, int imageW, int imageH,  int mask_w, int mask_h)
{
	const int ix = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const int iy = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

	if(ix < imageW && iy < imageH){
		
		float4 fresult = tex2D(texUCHAR, ix, iy);
		unsigned int min = make_color(fresult.x, fresult.y, fresult.z , 1.f);
		unsigned int new_min = 0;

		for (int m = ix - mask_w+1 ; m < ix + mask_w-1; m++){			
			for (int n = iy - mask_h+1 ; n < iy + mask_h-1; n++){				
				fresult = tex2D(texUCHAR, m, n);
				new_min = make_color(fresult.x, fresult.y, fresult.z , 1.f);
				if (min > new_min) min = new_min;					
			}
		}
		  

		dst[imageW * iy + ix] = min;
	
	}
}

extern "C" float grayErosionWrapper (unsigned int *dst, int imageW, int imageH, int threshold, int iteration, float brightness, float contrast,  int mask_w, int mask_h, int adjust)
{
	//for more effective kernel execution
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	unsigned int timer;
	float runtime;
	cutCreateTimer(&timer);
	cutStartTimer(timer);

	if(adjust)
		Grayscale<<<grid, threads>>>(dst, imageW, imageH, brightness, contrast);
	else
		Grayscale2<<<grid, threads>>>(dst, imageW, imageH);

	for(int i=0; i<iteration; i++)
	{
		hipMemcpyToArray( d_tempArray, 0, 0, dst, imageW * imageH * sizeof(unsigned int), hipMemcpyDeviceToDevice);
		hipBindTextureToArray(texUCHAR, d_tempArray);

		GrayErosion<<<grid, threads>>>(dst, imageW, imageH,  mask_w, mask_h);
	}	

	hipUnbindTexture(texUCHAR);
	hipDeviceSynchronize();
	cutStopTimer(timer);

	runtime = cutGetTimerValue(timer)/1000;
	cutDeleteTimer(timer);
	
	return runtime;
}