#include "hip/hip_runtime.h"
__global__ void Brightness (uint *dst, int imageW, int imageH, float brightness)
{
	const int ix = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const int iy = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

	if(ix < imageW && iy < imageH){

		const float x = (float)ix + 0.5f;
		const float y = (float)iy + 0.5f;

		float4 fresult = tex2D(texImage, x, y);
		float red = fresult.x;
		float green = fresult.y;
		float blue = fresult.z;

		red = red * (1.f - brightness) + brightness;
		green = green * (1.f - brightness) + brightness;
		blue = blue * (1.f - brightness) + brightness;

		dst[imageW * iy + ix] =  make_color(red, green, blue, 1.f);
	}
}


// if gamma is 0..1 , the dark intensities are stretched up
// if gamma is 1..5 , the high intensities are stretched down

extern "C" void brightnessWrapper (uint *dst, int imageW, int imageH, float brightness)
{
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	Brightness<<<grid, threads>>>(dst, imageW, imageH, brightness);
}