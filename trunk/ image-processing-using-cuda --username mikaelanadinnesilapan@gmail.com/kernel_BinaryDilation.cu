#include "hip/hip_runtime.h"
__global__ void BinaryDilation (unsigned int *dst, int imageW, int imageH,  int mask_w, int mask_h)
{
	const int ix = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const int iy = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

	if(ix < imageW && iy < imageH){
		
		int match = 1;
		for (int m = ix - mask_w ; m < ix + mask_w && match; m++){
			for (int n = iy - mask_h ; n < iy + mask_h && match; n++){
				float4 fresult = tex2D(texUCHAR, m, n);
				if (fresult.x == 0.f && fresult.y == 0.f && fresult.z == 0.f )
					match = 0;
			}
		}
		  

		if(!match)
		dst[imageW * iy + ix] = make_color(0.f, 0.f, 0.f , 1.f);
		else
		dst[imageW * iy + ix] = make_color(1.f, 1.f, 1.f , 1.f);
	
	}
}

extern "C" float binaryDilationWrapper (unsigned int *dst, int imageW, int imageH, int threshold, int iteration, float brightness, float contrast,  int mask_w, int mask_h, int adjust)
{
	//for more effective kernel execution
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	unsigned int timer;
	float runtime;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
	
	if(adjust)
		Grayscale<<<grid, threads>>>(dst, imageW, imageH, brightness, contrast);
	else
		Grayscale2<<<grid, threads>>>(dst, imageW, imageH);

	Binarize<<<grid, threads>>>(dst, imageW, imageH, threshold);

	for(int i=0; i<iteration; i++)
	{
		hipMemcpyToArray( d_tempArray, 0, 0, dst, imageW * imageH * sizeof(unsigned int), hipMemcpyDeviceToDevice);
		hipBindTextureToArray(texUCHAR, d_tempArray);

		BinaryDilation<<<grid, threads>>>(dst, imageW, imageH,  mask_w, mask_h);
	}

	hipUnbindTexture(texUCHAR);

	hipDeviceSynchronize();
	cutStopTimer(timer);
	
	runtime = cutGetTimerValue(timer)/1000;
	cutDeleteTimer(timer);
	
	return runtime;
}